#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <ctype.h>  // For isalpha() and isdigit()

// Kernel to encrypt a 4-character password
__global__ void encryptPasswordKernel(const char *rawPassword, char *newPassword) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Global thread index

    if (idx == 0) newPassword[0] = rawPassword[0] + 3; // Transform 1st letter
    if (idx == 1) newPassword[1] = rawPassword[0] - 2; // Transform 1st letter
    if (idx == 2) newPassword[2] = rawPassword[0] + 1; // Transform 1st letter
    if (idx == 3) newPassword[3] = rawPassword[1] + 1; // Transform 2nd letter
    if (idx == 4) newPassword[4] = rawPassword[1] - 2; // Transform 2nd letter
    if (idx == 5) newPassword[5] = rawPassword[1] - 3; // Transform 2nd letter
    if (idx == 6) newPassword[6] = rawPassword[2] + 1; // Transform 1st number
    if (idx == 7) newPassword[7] = rawPassword[2] - 2; // Transform 1st number
    if (idx == 8) newPassword[8] = rawPassword[3] + 4; // Transform 2nd number
    if (idx == 9) newPassword[9] = rawPassword[3] - 3; // Transform 2nd number

    if (idx < 10) {
        // Wrapping logic for letters and numbers
        if (idx < 6) { // First 6 characters are letters
            if (newPassword[idx] > 'z') {
                newPassword[idx] = 'a' + (newPassword[idx] - 'z' - 1);
            } else if (newPassword[idx] < 'a') {
                newPassword[idx] = 'z' - ('a' - newPassword[idx] - 1);
            }
        } else { // Last 4 characters are numbers
            if (newPassword[idx] > '9') {
                newPassword[idx] = '0' + (newPassword[idx] - '9' - 1);
            } else if (newPassword[idx] < '0') {
                newPassword[idx] = '9' - ('0' - newPassword[idx] - 1);
            }
        }
    }

    if (idx == 10) newPassword[10] = '\0'; // Null-terminate the string
}

// Kernel to decrypt a 10-character password back to the original 4-character password
__global__ void decryptPasswordKernel(const char *encryptedPassword, char *decryptedPassword) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Global thread index

    if (idx == 0) decryptedPassword[0] = encryptedPassword[0] - 3; // Reverse 1st letter
    if (idx == 1) decryptedPassword[1] = encryptedPassword[1] + 2; // Reverse 1st letter
    if (idx == 2) decryptedPassword[2] = encryptedPassword[2] - 1; // Reverse 1st letter
    if (idx == 3) decryptedPassword[3] = encryptedPassword[3] - 1; // Reverse 2nd letter
    if (idx == 4) decryptedPassword[4] = encryptedPassword[4] + 2; // Reverse 2nd letter
    if (idx == 5) decryptedPassword[5] = encryptedPassword[5] + 3; // Reverse 2nd letter
    if (idx == 6) decryptedPassword[6] = encryptedPassword[6] - 1; // Reverse 1st number
    if (idx == 7) decryptedPassword[7] = encryptedPassword[7] + 2; // Reverse 1st number
    if (idx == 8) decryptedPassword[8] = encryptedPassword[8] - 4; // Reverse 2nd number
    if (idx == 9) decryptedPassword[9] = encryptedPassword[9] + 3; // Reverse 2nd number

    // Wrapping logic for letters and numbers
    if (idx < 6) { // First 6 characters are letters
        if (decryptedPassword[idx] > 'z') {
            decryptedPassword[idx] = 'a' + (decryptedPassword[idx] - 'z' - 1);
        } else if (decryptedPassword[idx] < 'a') {
            decryptedPassword[idx] = 'z' - ('a' - decryptedPassword[idx] - 1);
        }
    } else { // Last 4 characters are numbers
        if (decryptedPassword[idx] > '9') {
            decryptedPassword[idx] = '0' + (decryptedPassword[idx] - '9' - 1);
        } else if (decryptedPassword[idx] < '0') {
            decryptedPassword[idx] = '9' - ('0' - decryptedPassword[idx] - 1);
        }
    }

    if (idx == 10) decryptedPassword[10] = '\0'; // Null-terminate the string
}

// Function to validate the password format
int validatePassword(const char *password) {
    // Check the length
    if (strlen(password) != 4) return 0;

    // Check if first two characters are letters (a-z, A-Z)
    if (!isalpha(password[0]) || !isalpha(password[1])) return 0;

    // Check if last two characters are digits (0-9)
    if (!isdigit(password[2]) || !isdigit(password[3])) return 0;

    return 1; // Valid password format
}

int main() {
    // Variables for user input
    char rawPasswordHost[5]; // Password must be 4 characters + null terminator
    int blocksPerGrid, threadsPerBlock;

    // Get password input from the user with validation
    while (1) {
        printf("Enter a 4-character password (2 letters followed by 2 digits): ");
        scanf("%4s", rawPasswordHost);
        
        // Validate the password
        if (validatePassword(rawPasswordHost)) {
            break;
        } else {
            printf("Invalid password format! Please enter a valid password (2 letters followed by 2 digits).\n");
        }
    }

    // Get CUDA configuration input from the user
    printf("Enter the number of blocks per grid: ");
    scanf("%d", &blocksPerGrid);

    printf("Enter the number of threads per block: ");
    scanf("%d", &threadsPerBlock);

    // Host and device memory for the encrypted and decrypted passwords
    char newPasswordHost[11]; // Host memory to store encrypted password
    char decryptedPasswordHost[11]; // Host memory to store decrypted password (10 characters + null terminator)
    char *rawPasswordDevice, *newPasswordDevice, *decryptedPasswordDevice;

    // Allocate device memory
    hipMalloc((void **)&rawPasswordDevice, sizeof(char) * 5);
    hipMalloc((void **)&newPasswordDevice, sizeof(char) * 11);
    hipMalloc((void **)&decryptedPasswordDevice, sizeof(char) * 11);

    // Copy the raw password to device memory
    hipMemcpy(rawPasswordDevice, rawPasswordHost, sizeof(char) * 5, hipMemcpyHostToDevice);

    // Launch the kernel for encryption
    encryptPasswordKernel<<<blocksPerGrid, threadsPerBlock>>>(rawPasswordDevice, newPasswordDevice);

    // Copy the encrypted password back to host memory
    hipMemcpy(newPasswordHost, newPasswordDevice, sizeof(char) * 11, hipMemcpyDeviceToHost);

    // Print the encrypted password
    printf("Encrypted password: %s\n", newPasswordHost);

    // Notify user that decryption is starting
    printf("Now decrypting the encrypted password...\n");

    // Launch the kernel for decryption
    decryptPasswordKernel<<<blocksPerGrid, threadsPerBlock>>>(newPasswordDevice, decryptedPasswordDevice);

    // Copy the decrypted password back to host memory
    hipMemcpy(decryptedPasswordHost, decryptedPasswordDevice, sizeof(char) * 11, hipMemcpyDeviceToHost);

    // Extract the original 4-character password from the decrypted string
    char finalDecryptedPassword[5]; // 4 characters + null terminator
    finalDecryptedPassword[0] = decryptedPasswordHost[0];  // 1st letter from index 0
    finalDecryptedPassword[1] = decryptedPasswordHost[3];  // 2nd letter from index 3
    finalDecryptedPassword[2] = decryptedPasswordHost[6];  // 1st number from index 6
    finalDecryptedPassword[3] = decryptedPasswordHost[8];  // 2nd number from index 8
    finalDecryptedPassword[4] = '\0';

    // Print the decrypted password (corrected)
    printf("Decrypted password: %s\n", finalDecryptedPassword);

    // Compare decrypted password with the original one
    if (strcmp(finalDecryptedPassword, rawPasswordHost) == 0) {
        printf("The decrypted password matches the original password.\n");
    } else {
        printf("The decrypted password does not match the original password. Could not crack the password.\n");
    }

    // Free device memory
    hipFree(rawPasswordDevice);
    hipFree(newPasswordDevice);
    hipFree(decryptedPasswordDevice);

    return 0;
}
